
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 64
#define BLOCK_SIZE 16

// 錯誤檢查包裝器
inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void matrixMulGPU(const int *a, const int *b, int *c)
{
    // 計算全局索引
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    // 累加變數
    int sum = 0;
    
    if (row < N && col < N)  // 邊界檢查
    {
        // 矩陣乘法計算
        for (int k = 0; k < N; ++k)
        {
            sum += a[row * N + k] * b[k * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main()
{
    int *a, *b, *c_cpu, *c_gpu;
    size_t size = N * N * sizeof(int);
    
    // 分配統一記憶體
    checkCuda(hipMallocManaged(&a, size));
    checkCuda(hipMallocManaged(&b, size));
    checkCuda(hipMallocManaged(&c_cpu, size));
    checkCuda(hipMallocManaged(&c_gpu, size));
    
    // 初始化矩陣
    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
        {
            a[i*N + j] = i;
            b[i*N + j] = j+2;
            c_cpu[i*N + j] = c_gpu[i*N + j] = 0;
        }
    
    // 設定執行配置
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, 
                   (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // 啟動核函數
    matrixMulGPU<<<numBlocks, threadsPerBlock>>>(a, b, c_gpu);
    
    // 檢查啟動錯誤
    checkCuda(hipGetLastError());
    // 同步等待
    checkCuda(hipDeviceSynchronize());
    
    // 釋放記憶體
    checkCuda(hipFree(a));
    checkCuda(hipFree(b));
    checkCuda(hipFree(c_cpu));
    checkCuda(hipFree(c_gpu));
    
    return 0;
}